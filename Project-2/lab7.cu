#include "hip/hip_runtime.h"
#include "c_img.h"
#include <stdio.h>

typedef struct rgb_img img_t;

__global__ void brighten(float adj_factor, int height, int width, uint8_t* in, uint8_t* out)
{
    // threadIdx thread number (0-M)
    // blockDim is how many threads per block
    // blockIdx is block number (0-N)
    int index = threadIdx.x + blockDim.x * blockIdx.x; 

    if (index < height*width*3)
    {
        out[index] = ((float)in[index] * adj_factor) > 256.0f ? 
                        (uint8_t)255 : (uint8_t)((float)in[index] * adj_factor);
    }
}

int main(int argc, char* argv[])
{
    img_t *im, *out50, *out10, *out200, *out400, *out120;
    read_in_img(&im, "UofTPresidentMericGertler-smaller.bin");


    // create the images
    create_img(&out50, im->height, im->width);
    create_img(&out10, im->height, im->width);
    create_img(&out200, im->height, im->width);
    create_img(&out400, im->height, im->width);
    create_img(&out120, im->height, im->width);


    uint8_t *d_raster, *d_out50, *d_out10, *d_out200, *d_out400, *d_out120;
    // allocate memory 
    hipMalloc((void**)&d_raster, sizeof(uint8_t)*im->width*im->height*3);
    hipMalloc((void**)&d_out10, sizeof(uint8_t)*im->width*im->height*3);
    hipMalloc((void**)&d_out50, sizeof(uint8_t)*im->width*im->height*3);
    hipMalloc((void**)&d_out400, sizeof(uint8_t)*im->width*im->height*3);
    hipMalloc((void**)&d_out200, sizeof(uint8_t)*im->width*im->height*3);
    hipMalloc((void**)&d_out120, sizeof(uint8_t)*im->width*im->height*3);
    


    hipMemcpy(d_raster, im->raster, sizeof(uint8_t)*im->width*im->height*3, hipMemcpyHostToDevice);

    brighten <<<(im->height*im->width*3 + 1023)/1024,1024>>> (0.1f, im->height, im->width, d_raster, d_out10);
    brighten <<<(im->height*im->width*3 + 1023)/1024,1024>>> (4.0f, im->height, im->width, d_raster, d_out400);
    brighten <<<(im->height*im->width*3 + 1023)/1024,1024>>> (2.0f, im->height, im->width, d_raster, d_out200);
    brighten <<<(im->height*im->width*3 + 1023)/1024,1024>>> (1.2f, im->height, im->width, d_raster, d_out120);
    brighten <<<(im->height*im->width*3 + 1023)/1024,1024>>> (0.5f, im->height, im->width, d_raster, d_out50);

    //Blocks have threads. First number above is how many blocks you want to launch, second is number of threads
    // [012345]  [012345]  [012345]

    hipMemcpy(out10->raster, d_out10, sizeof(uint8_t)*im->width*im->height*3, hipMemcpyDeviceToHost);
    hipMemcpy(out50->raster, d_out50, sizeof(uint8_t)*im->width*im->height*3, hipMemcpyDeviceToHost);
    hipMemcpy(out400->raster, d_out400, sizeof(uint8_t)*im->width*im->height*3, hipMemcpyDeviceToHost);
    hipMemcpy(out200->raster, d_out200, sizeof(uint8_t)*im->width*im->height*3, hipMemcpyDeviceToHost);
    hipMemcpy(out120->raster, d_out120, sizeof(uint8_t)*im->width*im->height*3, hipMemcpyDeviceToHost);
    
    hipFree(d_raster);
    hipFree(d_out10);
    hipFree(d_out120);
    hipFree(d_out200);
    hipFree(d_out400);
    hipFree(d_out50);

    write_img(out10, "Level0.1.bin");
    write_img(out50, "Level0.5.bin");
    write_img(out120, "Level1.2.bin");
    write_img(out400, "Level4.0.bin");
    write_img(out200, "Level2.0.bin");

    destroy_image(im);
    destroy_image(out10);
    destroy_image(out50);
    destroy_image(out120);
    destroy_image(out200);
    destroy_image(out400);
}